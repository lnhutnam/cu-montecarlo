#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>
#include <hiprand.h>

// includes, project
#include <hip/hip_runtime_api.h> // helper functions for cuda

// CUDA global constants
__constant__ int a, b, c, thread_i;

// Kernel routine
__global__ void square_avg(float *d_i, float *d_o)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int first_index = index * thread_i;
    float result = 0;
    for (int iter = first_index; iter < first_index + thread_i; iter++)
    {
        result += a * d_i[iter] * d_i[iter] + b * d_i[iter] + c;
    }
    d_o[index] = result / thread_i;
}

int main(int argc, const char **argv)
{
    const int h_a = 3.0;
    const int h_b = 5.0;
    const int h_c = 4.0;

    const int h_io = 640000;
    const int h_co = 6400;
    const int h_thread_i = h_io / h_co;

    // Allocate memory on host and device
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(a), &h_a, sizeof(h_a)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b), &h_b, sizeof(h_b)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c), &h_c, sizeof(h_c)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(thread_i), &h_thread_i, sizeof(h_thread_i)));

    float *d_in, *d_out, *h_out;
    checkCudaErrors(hipMalloc((void **)&d_in, sizeof(float) * h_io));
    checkCudaErrors(hipMalloc((void **)&d_out, sizeof(float) * h_co));
    h_out = (float *)malloc(sizeof(float) * h_co);

    // Random number generation
    hiprandGenerator_t random_generator;
    checkCudaErrors(hiprandCreateGenerator(&random_generator, HIPRAND_RNG_PSEUDO_DEFAULT));
    checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(random_generator, 1234ULL));
    checkCudaErrors(hiprandGenerateNormal(random_generator, d_in, h_io, 0.0f, 1.0f));

    square_avg<<<200, 32>>>(d_in, d_out);

    checkCudaErrors(hipMemcpy(h_out, d_out, sizeof(float) * h_co, hipMemcpyDeviceToHost));

    float out_sum = 0;
    for (int i = 0; i < h_co; i++)
    {
        out_sum += h_out[i];
    }

    printf("Mean:%f\n\n", out_sum / static_cast<float>(h_co));

    checkCudaErrors(hipFree(d_in));
    checkCudaErrors(hipFree(d_out));
    free(h_out);
}